
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>

__device__ void one_point_cross(int* newIndividuals, int* d_pop, int start, int end, hiprandState* my_curandstate, int dimensions, int pop_size)
{
	int pos = hiprand(my_curandstate) % (dimensions - 1);
	int partner = hiprand(my_curandstate) % pop_size;
	int cont = 0, sum = 0;
	for (int i = start; i < start + pos; i++)
	{

		newIndividuals[cont] = d_pop[i];
		sum += d_pop[i];
		cont++;
	}
	for (int i = dimensions * partner + pos; i < dimensions * partner + dimensions - 1; i++)
	{
		newIndividuals[cont] = d_pop[i];
		sum += d_pop[i];
		cont++;
	}
	newIndividuals[cont] = sum;
	cont++;
	sum = 0;
	for (int i = dimensions * partner; i < dimensions * partner + pos; i++)
	{
		newIndividuals[cont] = d_pop[i];
		sum += d_pop[i];
		cont++;
	}
	for (int i = start + pos; i < end - 1; i++)
	{
		newIndividuals[cont] = d_pop[i];
		sum += d_pop[i];
		cont++;
	}
	newIndividuals[cont] = sum;
}

__device__ void mutation(int* newIndividuals, int* d_pop, int start, int end, hiprandState* my_curandstate, int dimensions)
{
	int pos = hiprand(my_curandstate) % (dimensions - 1);
	int cont = 0, sum = 0;
	for (int i = start; i < end - 1; i++) {
		if (i == start + pos)
			newIndividuals[cont] = 1 - d_pop[i];
		else
			newIndividuals[cont] = d_pop[i];

		sum += newIndividuals[cont];
		cont++;
	}
	newIndividuals[cont] = sum;
}

extern "C"
__global__ void applyOperators(double* d_operators_probabilites, int* d_pop, int dimensions, int operators_number, hiprandState* my_curandstate, int pop_size, int totalThreads)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int row = idx * (pop_size / totalThreads);
	if (pop_size % totalThreads != 0)
		row++;
	if (idx < totalThreads)
	{
		for (int j = row; j < row + (pop_size / totalThreads); j++)
		{
			int i;
			int start, end;
			//Select Operator
			double sum = 0.0;
			int cont = 0;
			float rand_number = hiprand_uniform(my_curandstate + idx);
			for (int i = row * operators_number; i < row * operators_number + operators_number; i++)
			{
				sum += d_operators_probabilites[i];
				if (rand_number < sum)
					break;
				cont++;
			}

			start = row * dimensions;
			end = start + dimensions;
			float reward = -1.0;
			//Cross
			int newIndividuals[2 * 1000];
			if (cont == 0)
			{
				one_point_cross(newIndividuals, d_pop, start, end, my_curandstate + idx, dimensions, pop_size);
				int cont_new;
				if (newIndividuals[dimensions - 1] > d_pop[end - 1] && newIndividuals[dimensions - 1] > newIndividuals[2 * dimensions - 1])
				{
					cont_new = 0;
					for (int i = start; i < end; i++)
					{
						d_pop[i] = newIndividuals[cont_new];
						cont_new++;
					}
					reward = 1.0;
				}
				else if (newIndividuals[2 * dimensions - 1] > d_pop[end - 1] && newIndividuals[2 * dimensions - 1] > newIndividuals[dimensions - 1])
				{
					cont_new = dimensions;
					for (int i = start; i < end; i++)
					{
						d_pop[i] = newIndividuals[cont_new];
						cont_new++;
					}
					reward = 1.0;
				}

			}
			//Mutation
			else if (cont == 1)
			{
				mutation(newIndividuals, d_pop, start, end, my_curandstate + idx, dimensions);
				if (newIndividuals[dimensions - 1] > d_pop[end - 1])
				{
					int cont_new = 0;
					for (int i = start; i < end; i++)
					{
						d_pop[i] = newIndividuals[cont_new];
						cont_new++;
					}
					reward = 1.0;
				}
			}

			//Apply reward

			float plus = hiprand_uniform(my_curandstate + idx);
			plus = 1.0 + (plus * reward);
			d_operators_probabilites[row * operators_number + cont] *= plus;

			//Normalizes
			float sumP = 0.0;
			for (int i = row * operators_number; i < row * operators_number + operators_number; i++)
			{
				sumP += d_operators_probabilites[i];
			}

			for (int i = row * operators_number; i < row * operators_number + operators_number; i++)
			{
				d_operators_probabilites[i] /= sumP;
			}
		}
	}
}

